#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << "\n"; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

template <
    const int SRAM_M,
    const int SRAM_N,
    const int SRAM_K,
    const int REG_M,
    const int REG_N,
    const int REG_K
>
__global__ void sgemm_reg_f32_kernel(float* a, float* b, float* c, int M, int N, int K) {
    __shared__ float smem_a[SRAM_M][SRAM_K];
    __shared__ float smem_b[SRAM_K][SRAM_N];

    float reg_c[REG_M][REG_N] = {0};

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int tid = ty * blockDim.x + tx;
    int threads_per_block = blockDim.x * blockDim.y;

    for (int k0 = 0; k0 < K; k0 += SRAM_K) {
        for (int i = tid; i < SRAM_M * SRAM_K; i += threads_per_block) {
            int row = i / SRAM_K;
            int col = i % SRAM_K;
            int g_row = by * SRAM_M + row;
            int g_col = k0 + col;
            smem_a[row][col] = (g_row < M && g_col < K) ? a[g_row * K + g_col] : 0.0f;
        }

        for (int i = tid; i < SRAM_K * SRAM_N; i += threads_per_block) {
            int row = i / SRAM_N;
            int col = i % SRAM_N;
            int g_row = k0 + row;
            int g_col = bx * SRAM_N + col;
            smem_b[row][col] = (g_row < K && g_col < N) ? b[g_row * N + g_col] : 0.0f;
        }

        __syncthreads();

        for (int kk = 0; kk < SRAM_K; kk += REG_K) {
            float reg_a[REG_M][REG_K];
            float reg_b[REG_K][REG_N];

            for (int m = 0; m < REG_M; ++m) {
                int row = ty * REG_M + m;
                for (int k = 0; k < REG_K; ++k) {
                    int col = kk + k;
                    reg_a[m][k] = (row < SRAM_M && col < SRAM_K) ? smem_a[row][col] : 0.0f;
                }
            }

            for (int k = 0; k < REG_K; ++k) {
                int row = kk + k;
                for (int n = 0; n < REG_N; ++n) {
                    int col = tx * REG_N + n;
                    reg_b[k][n] = (row < SRAM_K && col < SRAM_N) ? smem_b[row][col] : 0.0f;
                }
            }

            for (int m = 0; m < REG_M; ++m)
                for (int n = 0; n < REG_N; ++n)
                    for (int k = 0; k < REG_K; ++k)
                        reg_c[m][n] += reg_a[m][k] * reg_b[k][n];
        }

        __syncthreads();
    }

    for (int m = 0; m < REG_M; ++m) {
        int g_row = by * SRAM_M + ty * REG_M + m;
        if (g_row >= M) continue;
        for (int n = 0; n < REG_N; ++n) {
            int g_col = bx * SRAM_N + tx * REG_N + n;
            if (g_col >= N) continue;
            c[g_row * N + g_col] = reg_c[m][n];
        }
    }
}

// Simple CPU GEMM for validation
void cpu_gemm(const float* A, const float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; ++i)
        for (int j = 0; j < N; ++j) {
            float acc = 0.0f;
            for (int k = 0; k < K; ++k)
                acc += A[i * K + k] * B[k * N + j];
            C[i * N + j] = acc;
        }
}

// Error checking
float max_abs_diff(const float* a, const float* b, int size) {
    float max_diff = 0.0f;
    for (int i = 0; i < size; ++i)
        max_diff = fmax(max_diff, fabs(a[i] - b[i]));
    return max_diff;
}

int main() {
    constexpr int M = 256, N = 256, K = 256;
    constexpr int SRAM_M = 64, SRAM_N = 64, SRAM_K = 8;
    constexpr int REG_M = 4, REG_N = 4, REG_K = 4;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    std::vector<float> h_A(M * K), h_B(K * N), h_C(M * N), h_C_ref(M * N);

    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice));

    dim3 threads(SRAM_N / REG_N, SRAM_M / REG_M);
    dim3 blocks((N + SRAM_N - 1) / SRAM_N, (M + SRAM_M - 1) / SRAM_M);

    auto start = std::chrono::high_resolution_clock::now();
    sgemm_reg_f32_kernel<SRAM_M, SRAM_N, SRAM_K, REG_M, REG_N, REG_K><<<blocks, threads>>>(d_A, d_B, d_C, M, N, K);
    CHECK_CUDA(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, size_C, hipMemcpyDeviceToHost));

    cpu_gemm(h_A.data(), h_B.data(), h_C_ref.data(), M, N, K);

    float err = max_abs_diff(h_C.data(), h_C_ref.data(), M * N);
    std::cout << "Max absolute error: " << err << "\n";

    double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
    double gflops = 2.0 * M * N * K / (time_ms / 1e3) / 1e9;
    std::cout << "Time: " << time_ms << " ms, Performance: " << gflops << " GFLOPS\n";

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    return 0;
}
