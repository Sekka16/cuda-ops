#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    return 0;
}